#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <algorithm>
#include <numeric>
#include <sstream>

#include "mpi.h"
#include <nccl.h>

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if( err != hipSuccess ) {                        \
    printf("Test CUDA failure %s:%d '%s'\n",    \
        __FILE__,__LINE__,hipGetErrorString(err)); \
    return err;                           \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t res = cmd;                           \
  if (res != ncclSuccess) {                         \
    printf("Test NCCL failure %s:%d '%s'\n",    \
        __FILE__,__LINE__,ncclGetErrorString(res)); \
    return res;                           \
  }                                                 \
} while(0)

using namespace std;

void testBcast(ncclUniqueId id, ncclComm_t *comms, int n, int ngpus) {
    int size, rank;
    MPI_Comm_size(MPI_COMM_WORLD, &size); 
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    double *h_data = new double[n];
    double **d_data = new double*[ngpus];
    for (int i = 0; i < ngpus; i++) {
        // CUDACHECK(hipSetDevice(i));
        // CUDACHECK(hipMalloc(&d_data[i], n * sizeof(double)));
        hipSetDevice(i);
        hipMalloc(&d_data[i], n * sizeof(double));
    }

    if (rank == 0) {
        for (int i = 0; i < n; ++i) {
            h_data[i] = (double) i;
        }
        random_shuffle(h_data, h_data + n);
        hipMemcpy(d_data[0], h_data, n * sizeof(double), hipMemcpyHostToDevice);
    }

    // only works for ngpus=1
    double t1 = MPI_Wtime();
    ncclBroadcast(d_data[0], d_data[0], n * sizeof(double), ncclDouble, 0, comms[0], hipStreamDefault);
    hipDeviceSynchronize();
    double t2 = MPI_Wtime();
    double time = t2-t1;
    double bandwidth = ((double)(n) * sizeof(double)) / time;
    cout << "rank: " << rank << " time: " << time << " bw: " << bandwidth << "\n";
}

int main(int argc, char* argv[])
{
    if(argc < 3)
    {
        cout << "Please specify the number of vertices (data size) in thousands and number of gpus per node";
        return 0;
    }
    int n;
    int ngpus;
    n = atoi(argv[1]);
    ngpus = atoi(argv[2]);
    n *= 1000;

    // MPI_Comm squarerowcomm, squarecolcomm;
    // MPI_Comm tallrowcomm, tallcolcomm;
    // MPI_Comm widerowcomm, widecolcomm;

    int rank, nprocs;
    MPI_Init( 0, 0 );
    MPI_Comm_size( MPI_COMM_WORLD, &nprocs); 
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);

    ncclUniqueId ncclId;
    if (rank == 0) {
        NCCLCHECK(ncclGetUniqueId(&ncclId));
    }
    MPI_Bcast(&ncclId, sizeof(ncclId), MPI_BYTE, 0, MPI_COMM_WORLD);

    ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t) * ngpus);
    NCCLCHECK(ncclGroupStart());
    for (int i = 0; i < ngpus; i++) {
        CUDACHECK(hipSetDevice(i));
        NCCLCHECK(ncclCommInitRank(&comms[i], nprocs * ngpus, ncclId, rank * ngpus + i));
    }
    NCCLCHECK(ncclGroupEnd());

    testBcast(ncclId, comms, n, ngpus);
    
    // // First do square grid 
    // int grcols = (int)std::sqrt((float)nprocs); 
    // int grrows = grcols; 
    // 
    // int myproccol = rank % grcols; 
    // int myprocrow = rank / grcols; 
    // MPI_Comm_split( MPI_COMM_WORLD, myprocrow, rank, &squarerowcomm );  
    // processes with the same color are in the same new communicator 
    // MPI_Comm_split( MPI_COMM_WORLD, myproccol, rank, &squarecolcomm );
    // if(rank == 0) cout << "*** Processor row ***" << endl;
    // DoA2A(squarerowcomm, 32*n, squarecolcomm);
    // if(rank == 0) cout << "*** Processor column ***" << endl;
    // DoA2A(squarecolcomm, 32*n, squarerowcomm);
    // if(rank == 0) cout << "*** Processor row ***" << endl;
    // DoAG(squarerowcomm, n, squarecolcomm);
    // if(rank == 0) cout << "*** Processor column ***" << endl;
    // DoAG(squarecolcomm, n, squarerowcomm);
    //     

    // if(rank == 0)
    //     cout << "### TALL GRID ###" << endl;
    // // Now do tall grid
    // int tallgrcols = grcols / 2;
    // int tallgrrows = grrows * 2; 
    //     myproccol = rank % tallgrcols;
    // myprocrow = rank / tallgrcols;
    //     MPI_Comm_split( MPI_COMM_WORLD, myprocrow, rank, &tallrowcomm );
    //     MPI_Comm_split( MPI_COMM_WORLD, myproccol, rank, &tallcolcomm );
    // DoA2A(tallrowcomm, 32*n, tallcolcomm);
    // DoA2A(tallcolcomm, 32*n, tallrowcomm);
    // DoAG(tallrowcomm, n, tallcolcomm);
    // DoAG(tallcolcomm, n, tallrowcomm);

    // if(rank == 0)
    //     cout << "### WIDE GRID ###" << endl;
    // // Now do wide grid
    // int widegrcols = grcols * 2;
    // int widegrrows = grrows / 2; 
    //     myproccol = rank % widegrcols;
    // myprocrow = rank / widegrcols;
    //     MPI_Comm_split( MPI_COMM_WORLD, myprocrow, rank, &widerowcomm );
    //     MPI_Comm_split( MPI_COMM_WORLD, myproccol, rank, &widecolcomm );
    // DoA2A(widerowcomm, 32*n, widecolcomm);
    // DoA2A(widecolcomm, 32*n, widerowcomm);
    // DoAG(widerowcomm, n, widecolcomm);
    // DoAG(widecolcomm, n, widerowcomm);

    for (int i = 0; i < ngpus; i++) {
        NCCLCHECK(ncclCommDestroy(comms[i]));
    }
    free(comms);
    MPI_Finalize( );
    
    return 0;
}

