#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Layout.h>
#include <ATen/Parallel.h>
#include <ATen/SparseTensorImpl.h>
#include <ATen/NativeFunctions.h>
#include <ATen/InitialTensorOptions.h>
#include <ATen/SparseTensorUtils.h>

#include "hipsparse.h"

#include <pybind11/pybind11.h>

#include <THC/THCGeneral.hpp>

#include <torch/extension.h>

namespace py = pybind11;

using namespace at::sparse;

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
    }                                                                          \
}

#define CHECK_ERROR(str) \
    {hipDeviceSynchronize(); hipError_t err; err = hipGetLastError(); if(err!=0) {printf("ERROR %s:  %d %s\n", str, err, hipGetErrorString(err)); fflush(stdout);}}


at::Tensor expand_values_if_needed(const at::Tensor& values) {
    // expand
    if (values.dim() == 0) {
        // Mimic Numpy behavior here and treat it as a 1D tensor
        return values.expand({1});
    } else {
        return values;
    }
}

at::Tensor sparse_coo_tensor_gpu(const at::Tensor& indices, 
                                    const at::Tensor& values_, 
                                    at::ArrayRef<int64_t> size) {

    at::Tensor values = expand_values_if_needed(values_); 

    int64_t sparse_dim = indices.size(0);
    int64_t dense_dim = values.dim() - 1;

    return at::_sparse_coo_tensor_with_dims_and_tensors(
        sparse_dim, dense_dim, size, indices, values, values.options().layout(at::kSparse));
}

template<typename T>
void printCusparseDnMat(int64_t rows, int64_t cols, int64_t ld, T *values_dev) {
  T* values_host = new T[rows*cols];
  hipMemcpy(values_host, values_dev, rows*cols*sizeof(T), hipMemcpyDeviceToHost);
  for (int64_t row = 0; row < rows; row++) {
    for (int64_t col = 0; col < cols; col++) {
      // Cusparse dense matrices are stored in column-major order
      std::cout << values_host[col*rows+row] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << "  values: ";
  for (int64_t i = 0; i < rows*cols; i++) {
    std::cout << values_host[i] << " ";
  }
  std::cout << std::endl;
  std::cout << "  shape: " << rows << ", " << cols << std::endl;
  delete [] values_host;
}

template<typename T>
void printCusparseSpMat(int32_t rows, int32_t cols, int32_t nnz, int32_t *row_indices_dev,
                            int32_t *col_indices_dev, T *values_dev) {
  T* values_host = new T[nnz];
  int32_t* row_indices_host = new int32_t[nnz];
  int32_t* col_indices_host = new int32_t[nnz];
  hipMemcpy(values_host, values_dev, nnz*sizeof(T), hipMemcpyDeviceToHost);
  hipMemcpy(row_indices_host, row_indices_dev, nnz*sizeof(int32_t), hipMemcpyDeviceToHost);
  hipMemcpy(col_indices_host, col_indices_dev, nnz*sizeof(int32_t), hipMemcpyDeviceToHost);

  for (int64_t i = 0; i < nnz; i++) {
    std::cout << "(" << row_indices_host[i]
      << ", " << col_indices_host[i]
      << "): " << values_host[i] << std::endl;
  }
  std::cout << "  values: ";
  for (int64_t i = 0; i < nnz; i++) {
    std::cout << values_host[i] << " ";
  }
  std::cout << std::endl;
  std::cout << "  row_indices: ";
  for (int64_t i = 0; i < nnz; i++) {
    std::cout << row_indices_host[i] << " ";
  }
  std::cout << std::endl;
  std::cout << "  col_indices: ";
  for (int64_t i = 0; i < nnz; i++) {
    std::cout << col_indices_host[i] << " ";
  }
  std::cout << std::endl;
  std::cout << "  shape: " << rows << ", " << cols << std::endl;
  delete [] values_host;
  delete [] row_indices_host;
  delete [] col_indices_host;
}

// at::Tensor spmm_gpu(const at::Tensor& A_rowindices, 
void spmm_gpu(const at::Tensor& A_rowindices, 
                        const at::Tensor& A_colindices,
                        const at::Tensor& A_values, 
                        int32_t n,
                        int32_t m,
                        at::Tensor& B,
                        at::Tensor& C) {

    // hipsparseHandle_t handle;
    // CHECK_CUSPARSE(hipsparseCreate(&handle));
    auto state = at::globalContext().lazyInitCUDA();
    // auto handle = THCState_getCurrentSparseHandle(state);
    auto handle = at::cuda::getCurrentCUDASparseHandle();

    int nnz = A_values.size(0);

    int32_t *d_a_csrrows;
    
    hipMalloc(&d_a_csrrows, (n + 1) * sizeof(int32_t));
    CHECK_CUSPARSE(hipsparseXcoo2csr(handle, 
                                        A_rowindices.data<int>(), 
                                        nnz, 
                                        n, 
                                        d_a_csrrows, 
                                        HIPSPARSE_INDEX_BASE_ZERO));

    float alpha = 1;
    float beta = 1;
    // hipsparseMatDescr_t descrA;
    // hipsparseCreateMatDescr(&descrA);
    // hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL);
    // hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO);

    int32_t b_row = B.size(0);
    int32_t b_col = B.size(1);
    int32_t c_row = C.size(0);
    int32_t c_col = C.size(1);
    
    // // Row-major to column-major
    // C.t_();
    // C.set_data(C.contiguous());
    // C.set_data(C.view({c_row, c_col}));

    // Create sparse matrix for A
    hipsparseSpMatDescr_t a_cusparse;
    CHECK_CUSPARSE(hipsparseCreateCsr(&a_cusparse, // hipsparseSpMatDescr_t* spMatDescr,
                                          n,            // int64_t               rows,
                                          n,            // int64_t               cols,
                                          nnz,          // int64_t               nnz,
                                          d_a_csrrows,  // void*                 csrRowOffsets,
                                          A_colindices.data<int>(), // void*                 csrColInd,
                                          A_values.data<float>(),   // void*                 csrValues,
                                          HIPSPARSE_INDEX_32I,       // hipsparseIndexType_t   csrRowOffsetsType,
                                          HIPSPARSE_INDEX_32I,       // hipsparseIndexType_t   csrColIndType,
                                          HIPSPARSE_INDEX_BASE_ZERO, // hipsparseIndexBase_t   idxBase,
                                          HIP_R_32F));             // hipDataType          valueType)
    
    // Create a dense matrix for B
    hipsparseDnMatDescr_t b_cusparse;
    CHECK_CUSPARSE(hipsparseCreateDnMat(&b_cusparse,     // hipsparseDnMatDescr_t* dnMatDescr,
                                            b_row,      // int64_t               rows,
                                            b_col,      // int64_t               cols,
                                            b_col,      // int64_t               ld,
                                            B.data<float>(),        // void*                 values,
                                            HIP_R_32F,             // hipDataType          valueType,
                                            HIPSPARSE_ORDER_ROW));   // hipsparseOrder_t       order)

    // Create a dense matrix for C
    hipsparseDnMatDescr_t c_cusparse;
    CHECK_CUSPARSE(hipsparseCreateDnMat(&c_cusparse,     // hipsparseDnMatDescr_t* dnMatDescr,
                                            c_row,      // int64_t               rows,
                                            c_col,      // int64_t               cols,
                                            c_col,      // int64_t               ld,
                                            C.data<float>(),        // void*                 values,
                                            HIP_R_32F,             // hipDataType          valueType,
                                            HIPSPARSE_ORDER_ROW));   // hipsparseOrder_t       order)

    // Compute external buffer size for spmm
    size_t bufferSize;
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, // hipsparseHandle_t     handle,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE, // hipsparseOperation_t  opA,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE, // hipsparseOperation_t  opB,
                                            &alpha,                             // const void*          alpha,
                                            a_cusparse,                         // hipsparseSpMatDescr_t matA,
                                            b_cusparse,                         // hipsparseDnMatDescr_t matB,
                                            &beta,                              // const void*          beta,
                                            c_cusparse,                         // hipsparseDnMatDescr_t matC,
                                            HIP_R_32F,                     // hipDataType         computeType,
                                            HIPSPARSE_SPMM_CSR_ALG2,         // hipsparseSpMMAlg_t    alg,
                                            &bufferSize));                  // size_t*              bufferSize)

    // Allocate external buffer for spmm
    char *externalBuffer = NULL;
    hipMalloc(&externalBuffer, bufferSize);
    
    // Run spmm 
    CHECK_CUSPARSE(hipsparseSpMM(handle, // hipsparseHandle_t     handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE, // hipsparseOperation_t  opA,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE, // hipsparseOperation_t  opB,
                                 &alpha,                                // const void*          alpha,
                                 a_cusparse,                            // hipsparseSpMatDescr_t matA,
                                 b_cusparse,                            // hipsparseDnMatDescr_t matB,
                                 &beta,                                 // const void*          beta,
                                 c_cusparse,                            // hipsparseDnMatDescr_t matC,
                                 HIP_R_32F,                            // hipDataType         computeType,
                                 HIPSPARSE_SPMM_CSR_ALG2,                // hipsparseSpMMAlg_t    alg,
                                 externalBuffer));                      //void*                externalBuffer)


    hipFree(d_a_csrrows);
    hipFree(externalBuffer);

    // // Column-major to row-major
    // // B.set_data(B.view({b_col, b_row}));
    // // B.t_();
    // C.set_data(C.view({c_col, c_row}));
    // C.t_();
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("sparse_coo_tensor_gpu", &sparse_coo_tensor_gpu, "Sparse Tensor GPU-only constructor");
    m.def("spmm_gpu", &spmm_gpu, "SpMM wrapper for cusparse");
}
