#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

int main(int argc, char** argv) {

  if(argc < 3) {
      std::cout << "Usage: ./nccl_ex_2d <datasize> <proccount>" << std::endl;
      return 0;
  }

  int n;
  int nprocs;
  n = atoi(argv[1]);
  nprocs = atoi(argv[2]);
  n *= 1000;

  int rank, size;

  // Initializing MPI
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &size));

  char hostname[1024];
  hostname[1023] = '\0';
  gethostname(hostname, 1023);
  printf("rank: %d, Hostname: %s\n", rank, hostname);

  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);

  int local_gpuid = rank % 3;
  // int local_gpuid = rank % 1;
  CUDACHECK(hipSetDevice(local_gpuid));

  // Obtain the group of processes in the world communicator
  MPI_Group world_group;
  MPI_Comm_group(MPI_COMM_WORLD, &world_group);

  // Remove all unnecessary ranks
  MPI_Group new_group;
  int ranges[1][3];
  ranges[0][0] = nprocs;
  ranges[0][1] = size - 1;
  ranges[0][2] = 1;
  int range_count = nprocs < size;
  
  MPI_Group_range_excl(world_group, range_count, ranges, &new_group);

  int procdim = (int)std::sqrt((float)nprocs);
  int proc_col_id = rank % procdim;
  int proc_row_id = rank / procdim;


  // Create a new communicator
  MPI_Comm mpi_new_world;
  MPI_Comm_create(MPI_COMM_WORLD, new_group, &mpi_new_world);

  if (mpi_new_world == MPI_COMM_NULL) {
    MPI_Finalize();
    exit(0);
  }

  // CUDACHECK(hipSetDevice(0));

  // float** sendbuff = (float**)malloc(ngpus * sizeof(float*));
  // float** recvbuff = (float**)malloc(ngpus * sizeof(float*));
  // hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*ngpus);
  // hipEvent_t start[ngpus];
  // hipEvent_t stop[ngpus];

  // float *h_data = new float[n]();
  double *h_data = new double[n]();
  for (int i = 0; i < n; i++) {
    h_data[i] = (double) (5.0);
  }

  // Initialize send/receive buffers, streams, and timers
  // float *sendbuff;
  // float *recvbuff;
  double *sendbuff;
  double *recvbuff;
  hipEvent_t start[3]; // row, col, overall
  hipEvent_t stop[3]; // row, col, overall

  CUDACHECK(hipMalloc(&sendbuff, n * sizeof(double)));
  CUDACHECK(hipMemcpy(sendbuff, h_data, n * sizeof(double), hipMemcpyHostToDevice));

  CUDACHECK(hipMalloc(&recvbuff, n * sizeof(double)));
  CUDACHECK(hipMemset(recvbuff, 0, n * sizeof(double)));

  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*2);
  CUDACHECK(hipStreamCreate(&s[0]));
  CUDACHECK(hipStreamCreate(&s[1]));

  CUDACHECK(hipEventCreate(&start[0]));
  CUDACHECK(hipEventCreate(&start[1]));
  CUDACHECK(hipEventCreate(&start[2]));
  CUDACHECK(hipEventCreate(&stop[0]));
  CUDACHECK(hipEventCreate(&stop[1]));
  CUDACHECK(hipEventCreate(&stop[2]));

  // Create 2D process grid in MPI to bcast NCCL unique id's
  MPI_Comm mpi_row_comm, mpi_col_comm;

  MPICHECK(MPI_Comm_split( mpi_new_world, proc_row_id, rank, &mpi_row_comm ));  // processes with the same color are in the same new communicator 
  MPICHECK(MPI_Comm_split( mpi_new_world, proc_col_id, rank, &mpi_col_comm ));

  // Create NCCL unique id's (one per subcommunicator) and MPI_bcast them to every proc in subcommunicator.
  ncclComm_t comms[2]; // one comm for row, one comm for col

  // Create NCCL unique id's for process rows
  ncclUniqueId id_row;

  // Generating NCCL unique ID at one process and broadcasting it to all
  if (proc_col_id == 0) {
    ncclGetUniqueId(&id_row);
  }

  MPICHECK(MPI_Bcast((void *)&id_row, sizeof(id_row), MPI_BYTE, 0, mpi_row_comm));

  MPI_Barrier(mpi_new_world);

  // Initialize NCCL communicator.
  NCCLCHECK(ncclCommInitRank(&comms[0], procdim, id_row, proc_col_id));

  MPI_Barrier(mpi_new_world);

  // Create NCCL unique id's for process cols
  ncclUniqueId id_col;

  // Generating NCCL unique ID at one process and broadcasting it to all
  if (proc_row_id == 0) {
    ncclGetUniqueId(&id_col);
  }

  MPICHECK(MPI_Bcast((void *)&id_col, sizeof(id_col), MPI_BYTE, 0, mpi_col_comm));

  MPI_Barrier(mpi_new_world);

  // Initialize NCCL communicator.
  NCCLCHECK(ncclCommInitRank(&comms[1], procdim, id_col, proc_row_id));

  MPI_Barrier(mpi_new_world);

  CUDACHECK(hipEventRecord(start[2], 0));
  
  // 2D SUMMA
  int trials = 20;
  float row_time = 0.0;
  float col_time = 0.0;

  for (int i = 0; i < procdim; i++) {

    CUDACHECK(hipEventRecord(start[0], 0));

    // Call ncclBroadcast (ncclGroup* calls make this function as one ncclBroadcast call).
    NCCLCHECK(ncclGroupStart());
    for (int j = 0; j < trials; j++) {
      NCCLCHECK(ncclBroadcast((const void*)sendbuff, (void*)recvbuff, n, ncclDouble, i, comms[0], 0));
    }
    NCCLCHECK(ncclGroupEnd());
    hipDeviceSynchronize();
    CUDACHECK(hipEventRecord(stop[0], 0));

    float row_bcast_time;
    CUDACHECK(hipEventElapsedTime(&row_bcast_time, start[0], stop[0]));
    CUDACHECK(hipEventSynchronize(stop[0]));
    row_time += row_bcast_time / 1000;

    CUDACHECK(hipEventRecord(start[1], 0));
    NCCLCHECK(ncclGroupStart());
    for (int j = 0; j < trials; j++) {
      NCCLCHECK(ncclBroadcast((const void*)sendbuff, (void*)recvbuff, n, ncclDouble, i, comms[1], 0));
    }
    NCCLCHECK(ncclGroupEnd());
    hipDeviceSynchronize();
    CUDACHECK(hipEventRecord(stop[1], 0));

    float col_bcast_time;
    CUDACHECK(hipEventElapsedTime(&col_bcast_time, start[1], stop[1]));
    CUDACHECK(hipEventSynchronize(stop[1]));
    col_time += col_bcast_time / 1000;
  }

  CUDACHECK(hipEventRecord(stop[2], 0));
  CUDACHECK(hipEventSynchronize(stop[0]));
  CUDACHECK(hipEventSynchronize(stop[1]));
  CUDACHECK(hipEventSynchronize(stop[2]));

  // Collect timings and verify broadcast worked.
  // float *h_recvbuff = new float[n]();
  double *h_recvbuff = new double[n]();
  float time_row, time_col, gpu_time;
  CUDACHECK(hipEventElapsedTime(&time_row, start[0], stop[0]));
  CUDACHECK(hipEventElapsedTime(&time_col, start[1], stop[1]));
  CUDACHECK(hipEventElapsedTime(&gpu_time, start[2], stop[2]));
  time_row /= 1000; // seconds
  time_col /= 1000; // seconds
  gpu_time /= 1000; // seconds

  hipMemcpy(h_recvbuff, recvbuff, n * sizeof(double), hipMemcpyDeviceToHost);
  for (int i = 0; i < n; i++) {
    if (h_recvbuff[i] != (double)(5.0)) {
      std::cout << "bcast error " << h_recvbuff[i] << " " << (procdim - 1) << std::endl;
      exit(0);
    }
  }

  std::cout << "rank: " << rank << " size: " << (n * sizeof(double)) << " gpu_time: " << gpu_time << " bw: " << ((2 * procdim * n * sizeof(double) * trials) / gpu_time) << std::endl;
  std::cout << "rank: " << rank << " size: " << (n * sizeof(double)) << " row_time: " << row_time << " bw: " << ((procdim * n * sizeof(double) * trials) / row_time) << std::endl;
  std::cout << "rank: " << rank << " size: " << (n * sizeof(double)) << " col_time: " << col_time << " bw: " << ((procdim * n * sizeof(double) * trials) / col_time) << std::endl;

  // Freeing device memory
  CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(recvbuff));

  // Finalizing NCCL
  ncclCommDestroy(comms[0]);
  ncclCommDestroy(comms[1]);

  // Finalizing MPI
  MPICHECK(MPI_Finalize());

  // printf("[MPI Rank %d] Success \n", rank);
  return 0;
}
